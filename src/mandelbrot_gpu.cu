#include "hip/hip_runtime.h"
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include "mandelbrot_gpu.h"

// Функция вычисления числа итераций для одной точки в устройстве
__device__ unsigned int num_iters(double cx, double cy, unsigned int max_iters) {
    thrust::complex<double> z(0.0, 0.0);
    thrust::complex<double> c(cx, cy);

    for (unsigned int i = 0; i <= max_iters; ++i) {
        if (thrust::abs(z) > 2.0) {
            return i;
        }
        z = z * z + c;
    }

    return max_iters;
}

// Kernel для выполнения вычислений для каждой точки параллельно
__global__ void mandelbrot_kernel(unsigned int* results, double* cx, double* cy, unsigned int max_iters, int num_points) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_points) {
        results[idx] = num_iters(cx[idx], cy[idx], max_iters);
    }
}

// Функция для вызова kernel и копирования данных
unsigned int* gpu_mandelbrot(double* cx, double* cy, size_t num_points, unsigned int max_iters) {
    unsigned int *dev_results, *results;
    double *dev_cx, *dev_cy;

    // Выделение памяти на хосте
    results = new unsigned int[num_points];

    // Выделение памяти на устройстве
    hipMalloc((void**)&dev_cx, num_points * sizeof(double));
    hipMalloc((void**)&dev_cy, num_points * sizeof(double));
    hipMalloc((void**)&dev_results, num_points * sizeof(unsigned int));

    // Копирование данных с хоста на устройство
    hipMemcpy(dev_cx, cx, num_points * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_cy, cy, num_points * sizeof(double), hipMemcpyHostToDevice);

    // Расчет количества блоков и потоков
    int threads_per_block = 256;
    int blocks_per_grid = (num_points + threads_per_block - 1) / threads_per_block;

    // Вызов kernel
    mandelbrot_kernel<<<blocks_per_grid, threads_per_block>>>(dev_results, dev_cx, dev_cy, max_iters, num_points);

    // Копирование результатов с устройства на хост
    hipMemcpy(results, dev_results, num_points * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Освобождение памяти на устройстве
    hipFree(dev_cx);
    hipFree(dev_cy);
    hipFree(dev_results);

    return results;
}
